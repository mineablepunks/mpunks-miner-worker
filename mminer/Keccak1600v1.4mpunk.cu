#include "hip/hip_runtime.h"
/*
	To use this source, cite the paper with the following bibtex:

	@inproceedings{DBLP:conf/crypto/SongLG17,
	author    = {Ling Song and Guohong Liao and Jian Guo},
	title     = {{Non-full Sbox Linearization: Applications to Collision Attacks on Round-Reduced Keccak}},
	booktitle = {Advances in Cryptology - {CRYPTO} 2017 - 37th Annual International Cryptology Conference, Santa Barbara, CA, USA, August 20-24, 2017, Proceedings, Part {II}},
	pages     = {428--451},
	year      = {2017},
	crossref  = {DBLP:conf/crypto/2017-2},
	url       = {https://doi.org/10.1007/978-3-319-63715-0_15},
	doi       = {10.1007/978-3-319-63715-0_15},
	timestamp = {Tue, 15 Aug 2017 07:01:19 +0200},
	biburl    = {http://dblp.org/rec/bib/conf/crypto/SongLG17},
	bibsource = {dblp computer science bibliography, http://dblp.org}
	}
	@proceedings{DBLP:conf/crypto/2017-2,
	editor    = {Jonathan Katz and Hovav Shacham},
	title     = {Advances in Cryptology - {CRYPTO} 2017 - 37th Annual International Cryptology Conference, Santa Barbara, CA, USA, August 20-24, 2017, Proceedings, Part {II}},
	series    = {Lecture Notes in Computer Science},
	volume    = {10402},
	publisher = {Springer},
	year      = {2017},
	url       = {https://doi.org/10.1007/978-3-319-63715-0},
	doi       = {10.1007/978-3-319-63715-0},
	isbn      = {978-3-319-63714-3},
	timestamp = {Mon, 14 Aug 2017 14:37:57 +0200},
	biburl    = {http://dblp.org/rec/bib/conf/crypto/2017-2},
	bibsource = {dblp computer science bibliography, http://dblp.org}
	}

	rewritten for mpunks @bxxd
*/

#include "mminer.h"

using namespace std;

__device__ uint64_t device_difficulty_upper = 0;
__device__ uint64_t device_difficulty_lower = 5731203885580;

texture<unsigned int, 1, hipReadModeElementType>
	texreference_input;

__constant__ uint64_t RC[24] = {
	0x0000000000000001, 0x0000000000008082, 0x800000000000808A,
	0x8000000080008000, 0x000000000000808B, 0x0000000080000001,
	0x8000000080008081, 0x8000000000008009, 0x000000000000008A,
	0x0000000000000088, 0x0000000080008009, 0x000000008000000A,
	0x000000008000808B, 0x800000000000008B, 0x8000000000008089,
	0x8000000000008003, 0x8000000000008002, 0x8000000000000080,
	0x000000000000800A, 0x800000008000000A, 0x8000000080008081,
	0x8000000000008080, 0x0000000080000001, 0x8000000080008008};

#define ROL(x, n) (((x) << (n)) | ((x) >> ((uint64_t)64 - (n))))

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg,
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

uint64_t rand_uint64(void)
{
	uint64_t r = 0;
	for (int i = 0; i < 64; i += 15 /*30*/)
	{
		r = r * ((uint64_t)RAND_MAX + 1) + rand();
	}
	return r;
}

//assume each inputs have the same input length

__device__ int device_hash_count = 0;
__device__ uint64_t device_found_nonce;

__global__ void Keccak1600(const int inputByte, uint8_t *output, const int outputByte, uint64_t startNonce)
{

	uint32_t num_keccak_blocks = inputByte / (DATA_BLOCK_SIZE << 1);

	uint64_t state00 = 0, state01 = 0, state02 = 0, state03 = 0, state04 = 0,
			 state10 = 0, state11 = 0, state12 = 0, state13 = 0, state14 = 0,
			 state20 = 0, state21 = 0, state22 = 0, state23 = 0, state24 = 0,
			 state30 = 0, state31 = 0, state32 = 0, state33 = 0, state34 = 0,
			 state40 = 0, state41 = 0, state42 = 0, state43 = 0, state44 = 0;
	uint64_t tmpState00 = 0, tmpState01 = 0, tmpState02 = 0, tmpState03 = 0, tmpState04 = 0,
			 tmpState10 = 0, tmpState11 = 0, tmpState12 = 0, tmpState13 = 0, tmpState14 = 0,
			 tmpState20 = 0, tmpState21 = 0, tmpState22 = 0, tmpState23 = 0, tmpState24 = 0,
			 tmpState30 = 0, tmpState31 = 0, tmpState32 = 0, tmpState33 = 0, tmpState34 = 0,
			 tmpState40 = 0, tmpState41 = 0, tmpState42 = 0, tmpState43 = 0, tmpState44 = 0;
	uint64_t Csum0, Csum1, Csum2, Csum3, Csum4, D0, D1, D2, D3, D4;

	uint64_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	uint64_t nonce = startNonce + thread;

	// nonce = startNonce + device_hash_count;

#if DEBUG
	// printf("nonce=%lu/0x%016x\n", nonce, nonce);
	printf("n=%lu t=%lu nk=%d bdim=%d bid=%d tid=%d\n", nonce, thread, num_keccak_blocks,
		   blockDim.x, blockIdx.x, threadIdx.x);
#else
	// printf("n=%lu t=%lu nk=%d bdim=%d bid=%d tid=%d\n", nonce, thread, num_keccak_blocks,
	// 	   blockDim.x, blockIdx.x, threadIdx.x);

	// if (nonce == 609667058559510631)
	// {
	// 	printf("here!!!!\n");
	// 	printf("n=%lu t=%lu nk=%d bdim=%d bid=%d tid=%d\n", nonce, thread, num_keccak_blocks,
	// 		   blockDim.x, blockIdx.x, threadIdx.x);
	// }
#endif

	uint64_t save_state00, save_state01, save_state02, save_state03;

	//absoring phase
	for (int k = 0; k < num_keccak_blocks; k++)
	{

#if 0 < DATA_BLOCK_SIZE
		// state00 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k];
		state00 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+1) << 32);
		// printf("%016llX\n", state00);
#endif

#if 1 < DATA_BLOCK_SIZE
		// state01 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+1];
		state01 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 2) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 3) << 32);

#endif

#if 2 < DATA_BLOCK_SIZE
		// state02 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+2];
		state02 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 4) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 5) << 32);

#endif

#if 3 < DATA_BLOCK_SIZE
		// state03 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+3];
		state03 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 6) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 7) << 32);

#endif

#if 4 < DATA_BLOCK_SIZE
		// state04 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+4];
		state04 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 8) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 9) << 32);

#endif

#if 5 < DATA_BLOCK_SIZE
		// state10 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+5];
		state10 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 10) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 11) << 32);

#endif

#if 6 < DATA_BLOCK_SIZE
		// state11 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+6];
		state11 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 12) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 13) << 32);

#endif

#if 7 < DATA_BLOCK_SIZE
		// state12 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+7];
		state12 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 14) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 15) << 32);

#endif

#if 8 < DATA_BLOCK_SIZE
		// state13 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+8];
		state13 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 16) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 17) << 32);

#endif

#if 9 < DATA_BLOCK_SIZE
		// state14 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+9];
		state14 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 18) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 19) << 32);

#endif

#if 10 < DATA_BLOCK_SIZE
		// state20 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+10];
		state20 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 20) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 21) << 32);

#endif

#if 11 < DATA_BLOCK_SIZE
		// state21 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+11];
		state21 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 22) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 23) << 32);

#endif

#if 12 < DATA_BLOCK_SIZE
		// state22 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+12];
		state22 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 24) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 25) << 32);

#endif

#if 13 < DATA_BLOCK_SIZE
		// state23 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+13];
		state23 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 26) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 27) << 32);

#endif

#if 14 < DATA_BLOCK_SIZE
		// state24 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+14];
		state24 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 28) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 29) << 32);

#endif

#if 15 < DATA_BLOCK_SIZE
		// state30 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+15];
		state30 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 30) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 31) << 32);

#endif

#if 16 < DATA_BLOCK_SIZE
		// state31 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+16];
		state31 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 32) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 33) << 32);

#endif

#if 17 < DATA_BLOCK_SIZE
		// state32 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+17];
		state32 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 34) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 35) << 32);

#endif

#if 18 < DATA_BLOCK_SIZE
		// state33 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+18];
		state33 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 36) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 37) << 32);

#endif

#if 19 < DATA_BLOCK_SIZE
		// state34 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+19];
		state34 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 38) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 39) << 32);

#endif

#if 20 < DATA_BLOCK_SIZE
		// state40 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+20];
		state40 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 40) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 41) << 32);

#endif

#if 21 < DATA_BLOCK_SIZE
		// state41 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+21];
		state41 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 42) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 43) << 32);

#endif

#if 22 < DATA_BLOCK_SIZE
		// state42 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+22];
		state42 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 44) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 45) << 32);

#endif

#if 23 < DATA_BLOCK_SIZE
		// state43 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+23];
		state43 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 46) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 47) << 32);

#endif

#if 24 < DATA_BLOCK_SIZE
		// state44 ^= input[(blockIdx.x*BLOCKX + threadIdx.x)*inputByte+ DATA_BLOCK_SIZE*k+24];
		state44 ^= (uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 48) ^ ((uint64_t)tex1Dfetch(texreference_input, (blockIdx.x * BLOCKX + threadIdx.x) * inputByte + DATA_BLOCK_SIZE * k + 49) << 32);
#endif

		state03 = cuda_swab64(nonce);

		save_state00 = cuda_swab64(state00);
		save_state01 = cuda_swab64(state01);
		save_state02 = cuda_swab64(state02);
		save_state03 = cuda_swab64(state03);
#if DEBUG
		printf("MSG:\n0x%016lx%016lx%016lx%016lx\n",
			   cuda_swab64(state00),
			   cuda_swab64(state01),
			   cuda_swab64(state02),
			   cuda_swab64(state03));
#endif
		// if (nonce == 609667058559510631)
		// {
		// 	printf("MSG:\n0x%016lx%016lx%016lx%016lx\n",
		// 		   cuda_swab64(state00),
		// 		   cuda_swab64(state01),
		// 		   cuda_swab64(state02),
		// 		   cuda_swab64(state03));
		// }

#pragma unroll 4
		for (int i = 0; i < Nr; i++)
		{
			Csum0 = state00 ^ state10 ^ state20 ^ state30 ^ state40;
			Csum1 = state01 ^ state11 ^ state21 ^ state31 ^ state41;
			Csum2 = state02 ^ state12 ^ state22 ^ state32 ^ state42;
			Csum3 = state03 ^ state13 ^ state23 ^ state33 ^ state43;
			Csum4 = state04 ^ state14 ^ state24 ^ state34 ^ state44;
			//
			D0 = Csum4 ^ ROL(Csum1, 1);
			D1 = Csum0 ^ ROL(Csum2, 1);
			D2 = Csum1 ^ ROL(Csum3, 1);
			D3 = Csum2 ^ ROL(Csum4, 1);
			D4 = Csum3 ^ ROL(Csum0, 1);

			state00 ^= D0;
			state01 ^= D1;
			state02 ^= D2;
			state03 ^= D3;
			state04 ^= D4;
			tmpState00 = state00;
			tmpState20 = ROL(state01, 1);
			tmpState40 = ROL(state02, 62);
			tmpState10 = ROL(state03, 28);
			tmpState30 = ROL(state04, 27);

			state10 ^= D0;
			state11 ^= D1;
			state12 ^= D2;
			state13 ^= D3;
			state14 ^= D4;

			tmpState31 = ROL(state10, 36);
			tmpState01 = ROL(state11, 44);
			tmpState21 = ROL(state12, 6);
			tmpState41 = ROL(state13, 55);
			tmpState11 = ROL(state14, 20);

			state20 ^= D0;
			state21 ^= D1;
			state22 ^= D2;
			state23 ^= D3;
			state24 ^= D4;

			tmpState12 = ROL(state20, 3);
			tmpState32 = ROL(state21, 10);
			tmpState02 = ROL(state22, 43);
			tmpState22 = ROL(state23, 25);
			tmpState42 = ROL(state24, 39);

			state30 ^= D0;
			state31 ^= D1;
			state32 ^= D2;
			state33 ^= D3;
			state34 ^= D4;

			tmpState43 = ROL(state30, 41);
			tmpState13 = ROL(state31, 45);
			tmpState33 = ROL(state32, 15);
			tmpState03 = ROL(state33, 21);
			tmpState23 = ROL(state34, 8);

			state40 ^= D0;
			state41 ^= D1;
			state42 ^= D2;
			state43 ^= D3;
			state44 ^= D4;

			//
			tmpState24 = ROL(state40, 18);
			tmpState44 = ROL(state41, 2);
			tmpState14 = ROL(state42, 61);
			tmpState34 = ROL(state43, 56);
			tmpState04 = ROL(state44, 14);

			//
			state00 = tmpState00 ^ ((~tmpState01) & tmpState02);
			state10 = tmpState10 ^ ((~tmpState11) & tmpState12);
			state20 = tmpState20 ^ ((~tmpState21) & tmpState22);
			state30 = tmpState30 ^ ((~tmpState31) & tmpState32);
			state40 = tmpState40 ^ ((~tmpState41) & tmpState42);

			state01 = tmpState01 ^ ((~tmpState02) & tmpState03);
			state11 = tmpState11 ^ ((~tmpState12) & tmpState13);
			state21 = tmpState21 ^ ((~tmpState22) & tmpState23);
			state31 = tmpState31 ^ ((~tmpState32) & tmpState33);
			state41 = tmpState41 ^ ((~tmpState42) & tmpState43);

			state02 = tmpState02 ^ ((~tmpState03) & tmpState04);
			state12 = tmpState12 ^ ((~tmpState13) & tmpState14);
			state22 = tmpState22 ^ ((~tmpState23) & tmpState24);
			state32 = tmpState32 ^ ((~tmpState33) & tmpState34);
			state42 = tmpState42 ^ ((~tmpState43) & tmpState44);

			state03 = tmpState03 ^ ((~tmpState04) & tmpState00);
			state13 = tmpState13 ^ ((~tmpState14) & tmpState10);
			state23 = tmpState23 ^ ((~tmpState24) & tmpState20);
			state33 = tmpState33 ^ ((~tmpState34) & tmpState30);
			state43 = tmpState43 ^ ((~tmpState44) & tmpState40);

			state04 = tmpState04 ^ ((~tmpState00) & tmpState01);
			state14 = tmpState14 ^ ((~tmpState10) & tmpState11);
			state24 = tmpState24 ^ ((~tmpState20) & tmpState21);
			state34 = tmpState34 ^ ((~tmpState30) & tmpState31);
			state44 = tmpState44 ^ ((~tmpState40) & tmpState41);

			state00 ^= RC[i];
		}
	}

	//squeezing phase;
#if 0 < HASH_SIZE
	memcpy(output+(blockIdx.x*BLOCKX + threadIdx.x)*HASH_SIZE, &state00, 8);
#endif

#if 8 < HASH_SIZE
	memcpy(output + (blockIdx.x * BLOCKX + threadIdx.x) * HASH_SIZE + 8, &state01, 8);
#endif

#if 16 < HASH_SIZE
	memcpy(output + (blockIdx.x * BLOCKX + threadIdx.x) * HASH_SIZE + 16, &state02, 8);
#endif

#if 24 < HASH_SIZE
	memcpy(output + (blockIdx.x * BLOCKX + threadIdx.x) * HASH_SIZE + 24, &state03, 8);
#endif

#if DEBUG
	// printf("state:0x%016lx\n", cuda_swab64(state00));
	printf("nonce=0x%016lx\nOUT: \n0x%016lx%016lx%016lx%016lx\n",
		   nonce,
		   cuda_swab64(state00),
		   cuda_swab64(state01),
		   cuda_swab64(state02),
		   cuda_swab64(state03));
#endif

	bool found = 0;
	uint32_t upper = 0;
	uint64_t lower = 0;

	lower = cuda_swab64(state03);
	upper = cuda_swab64(state02);
	upper = upper << 8;

	if (device_difficulty_upper && upper < device_difficulty_upper)
	{
		found = 1;
	}
	else
	{

		if (device_difficulty_upper == upper && lower < device_difficulty_lower)
		{
			found = 1;
		}
	}

	if (found)
	{

		// device_found_nonce = nonce;
		printf("IN: \n0x%016lx%016lx%016lx%016lx\n OUT: \n0x%016lx%016lx%016lx%016lx\n",
			   save_state00,
			   save_state01,
			   save_state02,
			   save_state03,
			   cuda_swab64(state00),
			   cuda_swab64(state01),
			   cuda_swab64(state02),
			   cuda_swab64(state03));
		printf(">>> FOUND XXX nonce=%lu/0x%016lx combined=0x%06lx%016lx difficulty=0x%06lx%016lx\n", nonce, nonce, upper, lower,
			   device_difficulty_upper, device_difficulty_lower);
	}

	atomicAdd(&device_hash_count, 1);

#if 32 < HASH_SIZE
	memcpy(output + (blockIdx.x * BLOCKX + threadIdx.x) * HASH_SIZE + 32, &state04, 8);
#endif
}

int Padding(uint8_t input[], int inputByte, uint8_t output[])
{
	int outputByte = R / 8 - (inputByte + 1) % (R / 8) + inputByte + 1;
	printf("Padding inputByte=%d outputByte=%d\n", inputByte, outputByte);
	memcpy(output, input, inputByte);
	memset(output + inputByte, 0, sizeof(uint8_t) * (outputByte - inputByte));
	output[inputByte] = SUFFIX;
	output[outputByte - 1] ^= 0x80;
	return outputByte;
}

//byte

// uint8_t m[] = {0x22, 0x23, 0x3E, 0x5F, 0xCC, 0x4E, 0xFC, 0x0E, 0xEB, 0x03, 0x0C, 0x72, 0xF9, 0x7A, 0x4E, 0x8A, 0x9D, 0xC4, 0xBB, 0x96, 0x18, 0x33, 0xDA, 0xE8, 0xEF, 0xED, 0xCF, 0xFD, 0xE2, 0xA3, 0xC0, 0x37, 0x00, 0x69, 0xCE, 0x65, 0xB3, 0x32, 0x38, 0xAC, 0x43, 0xD6, 0x47, 0x64, 0xFB, 0xDA, 0xDE, 0xDC, 0x6A, 0x22, 0xA3, 0x0C, 0x15, 0xCC, 0x01, 0x0D, 0x7F, 0xC3, 0xA4, 0x45, 0xE3, 0x5E, 0xDA, 0xB7, 0x69, 0x29, 0xD0, 0xAB, 0x6C, 0x48, 0x35, 0xF2, 0x1F, 0xA7, 0x2D, 0x20, 0xC3, 0x3E, 0x5F, 0xCC, 0x4E, 0xFC, 0x0E, 0xEB, 0x03, 0x0C, 0x72, 0xF9, 0x7A, 0x4E, 0x8A, 0x9D, 0xC4, 0xBB, 0x96, 0x18, 0x33, 0xDA, 0xE8, 0xEF, 0xED, 0xCF, 0xFD, 0xE2, 0xA3, 0xC0, 0x37, 0x00, 0x69, 0xCE, 0x65, 0xB3, 0x32, 0x38, 0xAC, 0x43, 0xD6, 0x47, 0x64, 0xFB, 0xDA, 0xDE, 0xDC};
// uint8_t msg[32] = {0x04, 0x22, 0x00, 0x00, 0x00, 0x00, 0x3B, 0x00, 0x19, 0x00, 0x00, 0x00,
// 				 0x7D, 0x43, 0x7E, 0x28, 0xCD, 0x73, 0xA3, 0xF4, 0x87,
// 				 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00};
uint8_t msg[32] = {0};

uint8_t output[BLOCKNUM * BLOCKX][HASH_SIZE];
uint8_t input[BLOCKSIZE];
uint8_t host_input[SUMDATASIZE];

// #define STREAMNUM 5 xxx

hipStream_t stream[STREAMNUM];
uint32_t *device_input[STREAMNUM];
uint8_t *device_output[STREAMNUM];

uint64_t getTime(void)
{
	uint64_t val = 0;
	struct timeval tv;

	gettimeofday(&tv, NULL);

	val = (((uint64_t)tv.tv_sec) * 1000 + ((uint64_t)tv.tv_usec) / 1000);
	// printf("getTime tv.tv_sec %ld tv_usec %ld val %ld\n", tv.tv_sec, tv.tv_usec, val);
	return (uint64_t)val;
}

void printMsg(const char *title, uint8_t *msg, int len)
{
	if (title)
	{
		printf("%s:\n0x", title);
	}
	else
	{
		printf("0x");
	}
	for (int i = 0; i < len; i++)
	{
		printf("%02X", msg[i]);
	}
	printf("\n");
}

void init(OPTS *opts)
{

	/* xxx random number */
	time_t t;
	srand((unsigned)time(&t));
	// mpz_init(rnum);
	// gmp_randinit_mt(rstate);
	// gmp_randseed_ui(rstate, rand());

	/* big nums */
	// mpz_init2(hash_mpz, 88);
	// gmp_printf("hash_mpz=%Zd\n", hash_mpz);
	// mpz_init_set_str(difficultyTarget_mpz, DIFFICULTY, 10);
	// gmp_printf("difficultyTarget_mpz=%Zd\n", difficultyTarget_mpz);

	const char *val;
	int base;
	mpz_t sender_mpz;
	mpz_t lastMinedPunkAsset_mpz;
	mpz_t difficulty_mpz;
	mpz_t startNonce_mpz;
	size_t count;

	if (opts->str_address)
	{
		val = opts->str_address;
	}
	else
	{
		val = DEFAULT_ADDRESS;
	}
	if (val && val[0] == '0' and val[1] == 'x')
	{
		val = val + 2;
		base = 16;
	}
	else
	{
		base = 10;
	}
	mpz_init_set_str(sender_mpz, &val[22], 16);
	gmp_printf("sender_mpz=%Zd/%018Zx\n", sender_mpz, sender_mpz);

	if (opts->str_lastMined)
	{
		val = opts->str_lastMined;
	}
	else
	{
		val = DEFAULT_LASTMINED;
	}
	if (val && val[0] == '0' and val[1] == 'x')
	{
		val = val + 2;
		base = 16;
	}
	else
	{
		base = 10;
	}

	mpz_init_set_str(lastMinedPunkAsset_mpz, val, base);
	gmp_printf("lastMinedPunkAsset_mpz=%Zd/0x%Zx\n", lastMinedPunkAsset_mpz, lastMinedPunkAsset_mpz);

	if (opts->str_startNonce)
	{
		val = opts->str_startNonce;
	}
	else
	{
		val = NULL;
	}

	if (val && val[0] == '0' and val[1] == 'x')
	{
		val = val + 2;
		base = 16;
	}
	else
	{
		base = 10;
	}

	if (val)
	{
		mpz_init_set_str(startNonce_mpz, val, base);
		gmp_printf("startNonce_mpz=%Zd/0x%Zx\n", startNonce_mpz, startNonce_mpz);
		mpz_export(&opts->startNonce, &count, 1, sizeof(opts->startNonce), 0, 0, startNonce_mpz);
	}

	if (opts->str_difficulty)
	{
		val = opts->str_difficulty;
	}
	else
	{
		val = DEFAULT_DIFFICULTY;
	}

	if (val && val[0] == '0' and val[1] == 'x')
	{
		val = val + 2;
		base = 16;
	}
	else
	{
		base = 10;
	}

	uint8_t difficulty[16];
	if (val)
	{
		mpz_init_set_str(difficulty_mpz, val, base);
		gmp_printf("difficulty_mpz=%Zd/0x%032Zx\n", difficulty_mpz, difficulty_mpz);
		mpz_export(difficulty, &count, 1, sizeof(difficulty), 0, 0, difficulty_mpz);
	}
	// printMsg("difficulty", difficulty, 16);
	opts->upper_difficulty = ((uint64_t *)difficulty)[1];
	opts->lower_difficulty = ((uint64_t *)difficulty)[0];
	// printf("0x%016lx %016lx\n", opts->upper_difficulty, opts->lower_difficulty);

	/* set msg */
	printMsg("pre msg", msg, 32);
	mpz_export(msg, &count, 1, 12, 1, 0, lastMinedPunkAsset_mpz);
	mpz_export(msg + 12, &count, 1, 9, 1, 0, sender_mpz);
	printMsg("pos msg", msg, 32);

	Padding(msg, sizeof(msg), input);
	for (int i = 0; i < STREAMNUM; i++)
	{
		hipStreamCreate(&stream[i]);
	}
	checkCUDAError("create stream error");
	printf("init.. writing %d blocks size_t=%d\n", BLOCKX * BLOCKNUM, BLOCKSIZE);
	for (int i = 0; i < BLOCKX * BLOCKNUM; i++)
	{
		memcpy(host_input + i * BLOCKSIZE, input, BLOCKSIZE);
		// printMsg("msg",host_input + i*BLOCKSIZE, 32);

		// break;
	}
}

void GetCudaMalloc(int length)
{
	for (int i = 0; i < STREAMNUM; i++)
	{
		hipMalloc(&device_input[i], BLOCKNUM * BLOCKX * BLOCKSIZE);
		checkCUDAError("malloc for device_input");
		hipMalloc(&device_output[i], BLOCKX * BLOCKNUM * HASH_SIZE);
		checkCUDAError("malloc for device_output");
	}
}

void FreeAll()
{
	printf("freeAll..\n");
	hipDeviceSynchronize();
	for (int i = 0; i < STREAMNUM; i++)
	{
		hipStreamDestroy(stream[i]);
		hipFree(device_input[i]);
		hipFree(device_output[i]);
	}
}

static int destructing = 0;
void destruct()
{
	printf("destruct..\n");
	if (destructing)
	{
		return;
	}
	FreeAll();
}

/* Signal Handler for SIGINT */
void sigintHandler(int sig_num)
{
	printf("caught signal: SIGINT\n");

	destruct();
}

/* Signal Handler for SIGTERM */
void sigtermHandler(int sig_num)
{
	printf("caught signal: SIGTERM\n");
	destruct();
}

void get_options(int argc, char **argv, OPTS *opts)
{
	int c;

	memset(opts, 0, sizeof(OPTS));

	static struct option long_options[] =
		{
			{"address", required_argument, 0, 'a'},
			{"difficulty", required_argument, 0, 'd'},
			{"startNonce", required_argument, 0, 's'},
			{"lastMined", required_argument, 0, 'l'},
			{"cudaDevice", required_argument, 0, 'x'},
			{"cudaDevice", no_argument, 0, 't'},
			{0, 0, 0, 0}};

	while (1)
	{
		int option_index = 0;

		c = getopt_long(argc, argv, "a:d:s:l:x:t", long_options, &option_index);

		/* Detect the end of the options. */
		if (c == -1)
			break;

		switch (c)
		{
		case '0':
			printf("have 0\n");
			break;
		case 'a':
			opts->str_address = strdup(optarg);
			printf("opt address='%s'\n", opts->str_address);
			break;
		case 'd':
			opts->str_difficulty = strdup(optarg);
			printf("opt difficulty='%s'\n", opts->str_difficulty);
			break;
		case 's':
			opts->str_startNonce = strdup(optarg);
			printf("opt startNonce='%s'\n", opts->str_startNonce);
			break;
		case 'l':
			opts->str_lastMined = strdup(optarg);
			printf("opt lastMined='%s'\n", opts->str_lastMined);
			break;
		case 'x':
			opts->device = atoi(optarg);
			printf("opt device='%d'\n", opts->device);
			break;
		case 't':
			opts->test = true;
			printf("opt test only\n");
			break;
		default:
			printf("option `%c` is not supported.\n", c);
			exit(0);
		}
	}
}

#define TESTROUND 100

int main(int argc, char **argv)
{
	printf("Hi There!!\n");

	OPTS opts;
	get_options(argc, argv, &opts);

	printf("using device %d\n.", opts.device);
	hipSetDevice(opts.device);
	checkCUDAError("set device");

	init(&opts);

	if (opts.test)
	{
		return 0;
	}
	GetCudaMalloc(BLOCKSIZE);

	timeval tpstart;
	// timeval tpend;
	double timeuse;
	printf("CUDA start\n");
	int cur = 0;
	gettimeofday(&tpstart, NULL);
	double all_sec = 0;
	uint64_t start = getTime();
	uint64_t tstart = start;
	uint64_t elapsed = 0;
	int n_hashes = 0;
	int hash_count = 0;

	signal(SIGINT, sigintHandler);
	signal(SIGTERM, sigtermHandler);

	time_t t;
	uint64_t found_nonce = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(device_found_nonce), &found_nonce, sizeof(found_nonce), 0, hipMemcpyHostToDevice);

	// startNonce = rand_uint64();

	// for (cur = 0; cur < STREAMNUM; cur++)
	// {
	// 	hipMemcpyAsync(device_input[cur], host_input, SUMDATASIZE, hipMemcpyHostToDevice, stream[cur]);
	// 	checkCUDAError("memcpy from buf to device_input");
	// }

	uint64_t startNonce;
	int run = 0;
#if DEBUG
	if (opts.str_startNonce)
	{
		startNonce = opts.startNonce;
	}
	else
	{
		startNonce = 609667058559510630;
	}

	for (int i = 0; i < 1; i++)
#else
	if (opts.str_startNonce)
	{
		startNonce = opts.startNonce;
	}
	else
	{
		startNonce = rand_uint64();
	}
	// startNonce = 609667058559510630;
	while (!destructing)
	// for (int i = 0; i < 2; i++)
#endif
	{

		printf("%s run=%d startNonce=%lu/0x%016lx ->>\n", ctime(&t), run, startNonce, startNonce);
		time(&t);

		hipMemcpyAsync(device_input[cur], host_input, SUMDATASIZE, hipMemcpyHostToDevice, stream[cur]);
		checkCUDAError("memcpy from buf to device_input");
		hipBindTexture(0, texreference_input, device_input[cur], SUMDATASIZE);

		hipMemcpyToSymbol(HIP_SYMBOL(device_difficulty_lower), &opts.lower_difficulty, sizeof(opts.lower_difficulty), 0, hipMemcpyHostToDevice);
		checkCUDAError("copy to symbol");
		hipMemcpyToSymbol(HIP_SYMBOL(device_difficulty_upper), &opts.upper_difficulty, sizeof(opts.upper_difficulty), 0, hipMemcpyHostToDevice);
		checkCUDAError("copy to symbol");

		hipEvent_t start, stop;
		float elapsedTime = 0.0;

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		Keccak1600<<<BLOCKNUM, BLOCKX, 0, stream[cur]>>>(BLOCKSIZE / 4, device_output[cur], HASH_SIZE, startNonce);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipMemcpyFromSymbol(&hash_count, HIP_SYMBOL(device_hash_count), sizeof(hash_count), 0, hipMemcpyDeviceToHost);
		// hipMemcpyFromSymbol(&found_nonce, HIP_SYMBOL(device_found_nonce), sizeof(found_nonce), 0, hipMemcpyDeviceToHost);

		if (found_nonce)
		{
			printf(">>>>>>>>>>>found_nonce=%lu\n", found_nonce);
			found_nonce = 0;
			hipMemcpyToSymbol(HIP_SYMBOL(device_found_nonce), &found_nonce, sizeof(found_nonce), 0, hipMemcpyHostToDevice);
		}

		hipEventElapsedTime(&elapsedTime, start, stop);

		printf("elapsedTime=%.2fms\n", elapsedTime);
		if (run != 0) // first time to call kernel will cost more second
			all_sec += elapsedTime;
		hipEventDestroy(start);
		hipEventDestroy(stop);
		// printf("copying %d bytes to output\n", HASH_SIZE * BLOCKNUM * BLOCKX);
		// hipMemcpyAsync(output, device_output[cur], HASH_SIZE * BLOCKNUM * BLOCKX, hipMemcpyDeviceToHost, stream[cur]);

		// checkOutput();

		// n_hashes += BLOCKX * BLOCKNUM;

		printf("hash_count=%d n_hashes=%d\n", hash_count, n_hashes);
		startNonce += hash_count;
		n_hashes += hash_count;
		hash_count = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(device_hash_count), &hash_count, sizeof(hash_count), 0, hipMemcpyHostToDevice);

		elapsed = getTime() - tstart;
		if (elapsed > 1000)
		{
			printf(">>> STATS.. nhashes=%lu/s\n", (n_hashes / elapsed) * 1000);
			n_hashes = 0;
			tstart = getTime();
		}

		cur = (cur + 1) % STREAMNUM;
		hipUnbindTexture(&texreference_input);
		run++;
		fflush(stdout);
	}
	hipDeviceSynchronize();
	// gettimeofday(&tpend, NULL);
	timeuse = all_sec * 1000; //1000000*(tpend.tv_sec-tpstart.tv_sec) + tpend.tv_usec-tpstart.tv_usec;
	printf("used time: %f s\n", timeuse / 1000000);
	// printf("blockpersecond 2^%.5f\n", log2(BLOCKNUM * BLOCKX * (TESTROUND - 1) / (timeuse / 1000000)));
	// printf("total block: %d\n", BLOCKNUM * BLOCKX * (TESTROUND - 1));
	checkCUDAError("kernel invocation");

	destruct();
	printf("END\n");
	// cout << sizeof(output) << endl;
	// for (int d = 0; d < 3; d++)
	// {
	// 	printf("block %d:\n", d);
	// 	printMsg(NULL, output[d], HASH_SIZE);
	// 	printMsg("src", host_input + d * BLOCKSIZE, 32);
	// }

	return 0;
}
